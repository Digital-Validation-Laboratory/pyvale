// system headers
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

// custom headers
#include "define.hpp"


namespace curandom {

    double *pixel_samples;
    hiprandGenerator_t rand_generator;
    int seed = 1;

    void destroy_generator(){
        hiprandDestroyGenerator(rand_generator);
        INFO_OUT("Destroying Cuda random number generator: ", "success");
    }
    
    
    void setup_curand_generator(){
        CURAND_CALL(hiprandCreateGenerator(&rand_generator, HIPRAND_RNG_PSEUDO_MTGP32));
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(rand_generator, seed));
        INFO_OUT("Curand Seed: ",seed);
        atexit(destroy_generator);
    }

    
    void generate_pixel_samples(int num_samples){
        // generate uniformly distributed floats
        CURAND_CALL(hiprandGenerateUniformDouble(rand_generator, pixel_samples, num_samples));
    }

}
